#include "hip/hip_runtime.h"

#include "LSH_CUDA_UPGRADE.cuh"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define ROTL(x,r)   ((x) << (r)) | ((x) >> (32-r)) 
#define ROTR(x,r)   ((x) >> (r)) | ((x) << (32-r)) 

__device__ static const UINT g_StepConstants[208] = {
   0x917caf90, 0x6c1b10a2, 0x6f352943, 0xcf778243, 0x2ceb7472, 0x29e96ff2, 0x8a9ba428, 0x2eeb2642,
   0x0e2c4021, 0x872bb30e, 0xa45e6cb2, 0x46f9c612, 0x185fe69e, 0x1359621b, 0x263fccb2, 0x1a116870,
   0x3a6c612f, 0xb2dec195, 0x02cb1f56, 0x40bfd858, 0x784684b6, 0x6cbb7d2e, 0x660c7ed8, 0x2b79d88a,
   0xa6cd9069, 0x91a05747, 0xcdea7558, 0x00983098, 0xbecb3b2e, 0x2838ab9a, 0x728b573e, 0xa55262b5,
   0x745dfa0f, 0x31f79ed8, 0xb85fce25, 0x98c8c898, 0x8a0669ec, 0x60e445c2, 0xfde295b0, 0xf7b5185a,
   0xd2580983, 0x29967709, 0x182df3dd, 0x61916130, 0x90705676, 0x452a0822, 0xe07846ad, 0xaccd7351,
   0x2a618d55, 0xc00d8032, 0x4621d0f5, 0xf2f29191, 0x00c6cd06, 0x6f322a67, 0x58bef48d, 0x7a40c4fd,
   0x8beee27f, 0xcd8db2f2, 0x67f2c63b, 0xe5842383, 0xc793d306, 0xa15c91d6, 0x17b381e5, 0xbb05c277,
   0x7ad1620a, 0x5b40a5bf, 0x5ab901a2, 0x69a7a768, 0x5b66d9cd, 0xfdee6877, 0xcb3566fc, 0xc0c83a32,
   0x4c336c84, 0x9be6651a, 0x13baa3fc, 0x114f0fd1, 0xc240a728, 0xec56e074, 0x009c63c7, 0x89026cf2,
   0x7f9ff0d0, 0x824b7fb5, 0xce5ea00f, 0x605ee0e2, 0x02e7cfea, 0x43375560, 0x9d002ac7, 0x8b6f5f7b,
   0x1f90c14f, 0xcdcb3537, 0x2cfeafdd, 0xbf3fc342, 0xeab7b9ec, 0x7a8cb5a3, 0x9d2af264, 0xfacedb06,
   0xb052106e, 0x99006d04, 0x2bae8d09, 0xff030601, 0xa271a6d6, 0x0742591d, 0xc81d5701, 0xc9a9e200,
   0x02627f1e, 0x996d719d, 0xda3b9634, 0x02090800, 0x14187d78, 0x499b7624, 0xe57458c9, 0x738be2c9,
   0x64e19d20, 0x06df0f36, 0x15d1cb0e, 0x0b110802, 0x2c95f58c, 0xe5119a6d, 0x59cd22ae, 0xff6eac3c,
   0x467ebd84, 0xe5ee453c, 0xe79cd923, 0x1c190a0d, 0xc28b81b8, 0xf6ac0852, 0x26efd107, 0x6e1ae93b,
   0xc53c41ca, 0xd4338221, 0x8475fd0a, 0x35231729, 0x4e0d3a7a, 0xa2b45b48, 0x16c0d82d, 0x890424a9,
   0x017e0c8f, 0x07b5a3f5, 0xfa73078e, 0x583a405e, 0x5b47b4c8, 0x570fa3ea, 0xd7990543, 0x8d28ce32,
   0x7f8a9b90, 0xbd5998fc, 0x6d7a9688, 0x927a9eb6, 0xa2fc7d23, 0x66b38e41, 0x709e491a, 0xb5f700bf,
   0x0a262c0f, 0x16f295b9, 0xe8111ef5, 0x0d195548, 0x9f79a0c5, 0x1a41cfa7, 0x0ee7638a, 0xacf7c074,
   0x30523b19, 0x09884ecf, 0xf93014dd, 0x266e9d55, 0x191a6664, 0x5c1176c1, 0xf64aed98, 0xa4b83520,
   0x828d5449, 0x91d71dd8, 0x2944f2d6, 0x950bf27b, 0x3380ca7d, 0x6d88381d, 0x4138868e, 0x5ced55c4,
   0x0fe19dcb, 0x68f4f669, 0x6e37c8ff, 0xa0fe6e10, 0xb44b47b0, 0xf5c0558a, 0x79bf14cf, 0x4a431a20,
   0xf17f68da, 0x5deb5fd1, 0xa600c86d, 0x9f6c7eb0, 0xff92f864, 0xb615e07f, 0x38d3e448, 0x8d5d3a6a,
   0x70e843cb, 0x494b312e, 0xa6c93613, 0x0beb2f4f, 0x928b5d63, 0xcbf66035, 0x0cb82c80, 0xea97a4f7,
   0x592c0f3b, 0x947c5f77, 0x6fff49b9, 0xf71a7e5a, 0x1de8c0f5, 0xc2569600, 0xc4e4ac8c, 0x823c9ce1
};

__device__ static inline void load_msg_blk(LSH_internal* i_state, const UINT* msgblk)
{
    i_state->submsg_e_l[0] = (msgblk[0]);
    i_state->submsg_e_l[1] = (msgblk[1]);
    i_state->submsg_e_l[2] = (msgblk[2]);
    i_state->submsg_e_l[3] = (msgblk[3]);
    i_state->submsg_e_l[4] = (msgblk[4]);
    i_state->submsg_e_l[5] = (msgblk[5]);
    i_state->submsg_e_l[6] = (msgblk[6]);
    i_state->submsg_e_l[7] = (msgblk[7]);
    i_state->submsg_e_r[0] = (msgblk[8]);
    i_state->submsg_e_r[1] = (msgblk[9]);
    i_state->submsg_e_r[2] = (msgblk[10]);
    i_state->submsg_e_r[3] = (msgblk[11]);
    i_state->submsg_e_r[4] = (msgblk[12]);
    i_state->submsg_e_r[5] = (msgblk[13]);
    i_state->submsg_e_r[6] = (msgblk[14]);
    i_state->submsg_e_r[7] = (msgblk[15]);
    i_state->submsg_o_l[0] = (msgblk[16]);
    i_state->submsg_o_l[1] = (msgblk[17]);
    i_state->submsg_o_l[2] = (msgblk[18]);
    i_state->submsg_o_l[3] = (msgblk[19]);
    i_state->submsg_o_l[4] = (msgblk[20]);
    i_state->submsg_o_l[5] = (msgblk[21]);
    i_state->submsg_o_l[6] = (msgblk[22]);
    i_state->submsg_o_l[7] = (msgblk[23]);
    i_state->submsg_o_r[0] = (msgblk[24]);
    i_state->submsg_o_r[1] = (msgblk[25]);
    i_state->submsg_o_r[2] = (msgblk[26]);
    i_state->submsg_o_r[3] = (msgblk[27]);
    i_state->submsg_o_r[4] = (msgblk[28]);
    i_state->submsg_o_r[5] = (msgblk[29]);
    i_state->submsg_o_r[6] = (msgblk[30]);
    i_state->submsg_o_r[7] = (msgblk[31]);
}

__device__ static void msg_exp_even(LSH_internal* i_state)
{
    UINT temp;
    temp = i_state->submsg_e_l[0];
    i_state->submsg_e_l[0] = i_state->submsg_o_l[0] + i_state->submsg_e_l[3];
    i_state->submsg_e_l[3] = i_state->submsg_o_l[3] + i_state->submsg_e_l[1];
    i_state->submsg_e_l[1] = i_state->submsg_o_l[1] + i_state->submsg_e_l[2];
    i_state->submsg_e_l[2] = i_state->submsg_o_l[2] + temp;
    temp = i_state->submsg_e_l[4];
    i_state->submsg_e_l[4] = i_state->submsg_o_l[4] + i_state->submsg_e_l[7];
    i_state->submsg_e_l[7] = i_state->submsg_o_l[7] + i_state->submsg_e_l[6];
    i_state->submsg_e_l[6] = i_state->submsg_o_l[6] + i_state->submsg_e_l[5];
    i_state->submsg_e_l[5] = i_state->submsg_o_l[5] + temp;
    temp = i_state->submsg_e_r[0];
    i_state->submsg_e_r[0] = i_state->submsg_o_r[0] + i_state->submsg_e_r[3];
    i_state->submsg_e_r[3] = i_state->submsg_o_r[3] + i_state->submsg_e_r[1];
    i_state->submsg_e_r[1] = i_state->submsg_o_r[1] + i_state->submsg_e_r[2];
    i_state->submsg_e_r[2] = i_state->submsg_o_r[2] + temp;
    temp = i_state->submsg_e_r[4];
    i_state->submsg_e_r[4] = i_state->submsg_o_r[4] + i_state->submsg_e_r[7];
    i_state->submsg_e_r[7] = i_state->submsg_o_r[7] + i_state->submsg_e_r[6];
    i_state->submsg_e_r[6] = i_state->submsg_o_r[6] + i_state->submsg_e_r[5];
    i_state->submsg_e_r[5] = i_state->submsg_o_r[5] + temp;
}

__device__ static void msg_exp_odd(LSH_internal* i_state)
{
    UINT temp;
    temp = i_state->submsg_o_l[0];
    i_state->submsg_o_l[0] = i_state->submsg_e_l[0] + i_state->submsg_o_l[3];
    i_state->submsg_o_l[3] = i_state->submsg_e_l[3] + i_state->submsg_o_l[1];
    i_state->submsg_o_l[1] = i_state->submsg_e_l[1] + i_state->submsg_o_l[2];
    i_state->submsg_o_l[2] = i_state->submsg_e_l[2] + temp;
    temp = i_state->submsg_o_l[4];
    i_state->submsg_o_l[4] = i_state->submsg_e_l[4] + i_state->submsg_o_l[7];
    i_state->submsg_o_l[7] = i_state->submsg_e_l[7] + i_state->submsg_o_l[6];
    i_state->submsg_o_l[6] = i_state->submsg_e_l[6] + i_state->submsg_o_l[5];
    i_state->submsg_o_l[5] = i_state->submsg_e_l[5] + temp;
    temp = i_state->submsg_o_r[0];
    i_state->submsg_o_r[0] = i_state->submsg_e_r[0] + i_state->submsg_o_r[3];
    i_state->submsg_o_r[3] = i_state->submsg_e_r[3] + i_state->submsg_o_r[1];
    i_state->submsg_o_r[1] = i_state->submsg_e_r[1] + i_state->submsg_o_r[2];
    i_state->submsg_o_r[2] = i_state->submsg_e_r[2] + temp;
    temp = i_state->submsg_o_r[4];
    i_state->submsg_o_r[4] = i_state->submsg_e_r[4] + i_state->submsg_o_r[7];
    i_state->submsg_o_r[7] = i_state->submsg_e_r[7] + i_state->submsg_o_r[6];
    i_state->submsg_o_r[6] = i_state->submsg_e_r[6] + i_state->submsg_o_r[5];
    i_state->submsg_o_r[5] = i_state->submsg_e_r[5] + temp;
}

__device__ static inline void load_sc(const UINT** p_const_v, UINT i)
{
    *p_const_v = &g_StepConstants[i];
}

__device__ static void msg_add_even(UINT* cv_l, UINT* cv_r, LSH_internal* i_state)
{
    cv_l[0] ^= i_state->submsg_e_l[0]; cv_l[1] ^= i_state->submsg_e_l[1]; cv_l[2] ^= i_state->submsg_e_l[2]; cv_l[3] ^= i_state->submsg_e_l[3];
    cv_l[4] ^= i_state->submsg_e_l[4]; cv_l[5] ^= i_state->submsg_e_l[5]; cv_l[6] ^= i_state->submsg_e_l[6]; cv_l[7] ^= i_state->submsg_e_l[7];
    cv_r[0] ^= i_state->submsg_e_r[0]; cv_r[1] ^= i_state->submsg_e_r[1]; cv_r[2] ^= i_state->submsg_e_r[2]; cv_r[3] ^= i_state->submsg_e_r[3];
    cv_r[4] ^= i_state->submsg_e_r[4]; cv_r[5] ^= i_state->submsg_e_r[5]; cv_r[6] ^= i_state->submsg_e_r[6]; cv_r[7] ^= i_state->submsg_e_r[7];
}
__device__ static void msg_add_odd(UINT* cv_l, UINT* cv_r, LSH_internal* i_state)
{
    cv_l[0] ^= i_state->submsg_o_l[0]; cv_l[1] ^= i_state->submsg_o_l[1]; cv_l[2] ^= i_state->submsg_o_l[2]; cv_l[3] ^= i_state->submsg_o_l[3];
    cv_l[4] ^= i_state->submsg_o_l[4]; cv_l[5] ^= i_state->submsg_o_l[5]; cv_l[6] ^= i_state->submsg_o_l[6]; cv_l[7] ^= i_state->submsg_o_l[7];
    cv_r[0] ^= i_state->submsg_o_r[0]; cv_r[1] ^= i_state->submsg_o_r[1]; cv_r[2] ^= i_state->submsg_o_r[2]; cv_r[3] ^= i_state->submsg_o_r[3];
    cv_r[4] ^= i_state->submsg_o_r[4]; cv_r[5] ^= i_state->submsg_o_r[5]; cv_r[6] ^= i_state->submsg_o_r[6]; cv_r[7] ^= i_state->submsg_o_r[7];
}

__device__ static void add_blk(UINT* cv_l, const UINT* cv_r)
{
    cv_l[0] += cv_r[0];
    cv_l[1] += cv_r[1];
    cv_l[2] += cv_r[2];
    cv_l[3] += cv_r[3];
    cv_l[4] += cv_r[4];
    cv_l[5] += cv_r[5];
    cv_l[6] += cv_r[6];
    cv_l[7] += cv_r[7];
}

__device__ static void rotate_blk(UINT cv[8], const int rot_value)
{
    cv[0] = ROTL(cv[0], rot_value);
    cv[1] = ROTL(cv[1], rot_value);
    cv[2] = ROTL(cv[2], rot_value);
    cv[3] = ROTL(cv[3], rot_value);
    cv[4] = ROTL(cv[4], rot_value);
    cv[5] = ROTL(cv[5], rot_value);
    cv[6] = ROTL(cv[6], rot_value);
    cv[7] = ROTL(cv[7], rot_value);
}

__device__ static void  xor_with_const(UINT* cv_l, const UINT* const_v)
{
    cv_l[0] ^= const_v[0];
    cv_l[1] ^= const_v[1];
    cv_l[2] ^= const_v[2];
    cv_l[3] ^= const_v[3];
    cv_l[4] ^= const_v[4];
    cv_l[5] ^= const_v[5];
    cv_l[6] ^= const_v[6];
    cv_l[7] ^= const_v[7];
}

__device__ static void rotate_msg_gamma(UINT* cv_r)
{
    cv_r[1] = ROTL(cv_r[1], 8);
    cv_r[2] = ROTL(cv_r[2], 16);
    cv_r[3] = ROTL(cv_r[3], 24);
    cv_r[4] = ROTL(cv_r[4], 24);
    cv_r[5] = ROTL(cv_r[5], 16);
    cv_r[6] = ROTL(cv_r[6], 8);
}

__device__ static void word_perm(UINT* cv_l, UINT* cv_r)
{
    UINT temp;
    temp = cv_l[0];
    cv_l[0] = cv_l[6];
    cv_l[6] = cv_r[6];
    cv_r[6] = cv_r[2];
    cv_r[2] = cv_l[1];
    cv_l[1] = cv_l[4];
    cv_l[4] = cv_r[4];
    cv_r[4] = cv_r[0];
    cv_r[0] = cv_l[2];
    cv_l[2] = cv_l[5];
    cv_l[5] = cv_r[7];
    cv_r[7] = cv_r[1];
    cv_r[1] = temp;
    temp = cv_l[3];
    cv_l[3] = cv_l[7];
    cv_l[7] = cv_r[5];
    cv_r[5] = cv_r[3];
    cv_r[3] = temp;
};

__device__ static void mix(UINT* cv_l, UINT* cv_r, const UINT* const_v, const int rot_alpha, const int rot_beta)
{
    add_blk(cv_l, cv_r);
    rotate_blk(cv_l, rot_alpha);
    xor_with_const(cv_l, const_v);
    add_blk(cv_r, cv_l);
    rotate_blk(cv_r, rot_beta);
    add_blk(cv_l, cv_r);
    rotate_msg_gamma(cv_r);
}

__device__ static void initial_vector(LSH_Info* ctx)
{
    ctx->uChainVar_left[0] = 0x46a10f1f;
    ctx->uChainVar_left[1] = 0xfddce486;
    ctx->uChainVar_left[2] = 0xb41443a8;
    ctx->uChainVar_left[3] = 0x198e6b9d;
    ctx->uChainVar_left[4] = 0x3304388d;
    ctx->uChainVar_left[5] = 0xb0f5a3c7;
    ctx->uChainVar_left[6] = 0xb36061c4;
    ctx->uChainVar_left[7] = 0x7adbd553;
    ctx->uChainVar_right[0] = 0x105d5378;
    ctx->uChainVar_right[1] = 0x2f74de54;
    ctx->uChainVar_right[2] = 0x5c2f2d95;
    ctx->uChainVar_right[3] = 0xf2553fbe;
    ctx->uChainVar_right[4] = 0x8051357a;
    ctx->uChainVar_right[5] = 0x138668c8;
    ctx->uChainVar_right[6] = 0x47aa4484;
    ctx->uChainVar_right[7] = 0xe01afb41;
}

__device__ static void final(LSH_Info* Info)
{
    UINT i;
    for (i = 0; i < 8; i++)
    {
        Info->uChainVar_left[i] = Info->uChainVar_left[i] ^ Info->uChainVar_right[i];
    }
}

__device__ void LSH_Compress(LSH_Info* ctx, UINT_PTR sv_pt)
{
    UINT i;
    LSH_internal i_state[1];

    const UINT* const_v = NULL;
    UINT* cv_l = ctx->uChainVar_left;
    UINT* cv_r = ctx->uChainVar_right;

    load_msg_blk(i_state, sv_pt);

    msg_add_even(cv_l, cv_r, i_state);
    load_sc(&const_v, 0);
    mix(cv_l, cv_r, const_v, 29, 1);
    word_perm(cv_l, cv_r);

    msg_add_odd(cv_l, cv_r, i_state);
    load_sc(&const_v, 8);
    mix(cv_l, cv_r, const_v, 5, 17);
    word_perm(cv_l, cv_r);

    for (i = 1; i < 26 / 2; i++)
    {
        msg_exp_even(i_state);
        msg_add_even(cv_l, cv_r, i_state);
        load_sc(&const_v, 16 * i);
        mix(cv_l, cv_r, const_v, 29, 1);
        word_perm(cv_l, cv_r);

        msg_exp_odd(i_state);
        msg_add_odd(cv_l, cv_r, i_state);
        load_sc(&const_v, 16 * i + 8);
        mix(cv_l, cv_r, const_v, 5, 17);
        word_perm(cv_l, cv_r);
    }

    msg_exp_even(i_state);
    msg_add_even(cv_l, cv_r, i_state);
}


__device__ void LSH_Init(LSH_Info* Info)
{
    Info->remain_byte_len = 0;

    initial_vector(Info);

    return;
}

__device__ void LSH_update(LSH_Info* Info, const BYTE* pt, UINT pt_byte_len)
{
    UINT i = 0, t = 0;

    UINT remain_pt_byte;
    UINT pt_len = pt_byte_len;

    BYTE TEST_SV_PT[TEST_PT_SIZE] = { 0 };

    if (pt_byte_len == 0)
    {
        return;
    }

    for (int i = 0; i < TEST_PT_SIZE; i++)
    {
        TEST_SV_PT[i] = pt[i * blockDim.x * gridDim.x];
    }

    remain_pt_byte = Info->remain_byte_len;

    if (pt_len + remain_pt_byte < LSH_BLOCK_LEN)
    {
        memcpy((UCHAR_PTR)Info->sv_last_pt + remain_pt_byte, TEST_SV_PT, pt_len);
        Info->remain_byte_len += (UINT)pt_byte_len;
        return;
    }

    while (pt_len + remain_pt_byte >= LSH_BLOCK_LEN)
    {
        memcpy((UCHAR_PTR)(Info->sv_pt), TEST_SV_PT + i * LSH_BLOCK_LEN, (int)LSH_BLOCK_LEN);
        LSH_Compress(Info, (UINT_PTR)Info->sv_pt);

        i++;
        pt_len -= (LSH_BLOCK_LEN - remain_pt_byte);
        remain_pt_byte = 0;
    }

    memcpy((UCHAR_PTR)Info->sv_last_pt, TEST_SV_PT + i * LSH_BLOCK_LEN, pt_len);
    Info->remain_byte_len = (UINT)pt_len;

    return;
}

__device__ void LSH_final(LSH_Info* Info, BYTE* sv_hashval)
{
    UINT remain_pt_byte;

    remain_pt_byte = Info->remain_byte_len;

    Info->sv_last_pt[remain_pt_byte] = 0x80;

    memset(Info->sv_last_pt + remain_pt_byte + 1, 0, LSH_BLOCK_LEN - remain_pt_byte - 1);

    LSH_Compress(Info, (UINT_PTR)Info->sv_last_pt);

    final(Info);

    memcpy(sv_hashval, Info->uChainVar_left, sizeof(BYTE) * 32);

    memset(Info, 0, sizeof(LSH_Info));

    return;
}

__global__ void make_hash_val(LSH_Info* Info, BYTE* pt, BYTE* sv_hashval)
{
    int tid;
    tid = threadIdx.x + blockIdx.x * blockDim.x;

    LSH_Info us_Info[1];
    memcpy(us_Info, Info, sizeof(LSH_Info));

    LSH_Init(us_Info);
    LSH_update(us_Info, pt + tid, TEST_PT_SIZE);
    LSH_final(us_Info, sv_hashval + (tid * LSH_HASH_LEN));
}

void test_LSH_GPU(ULL Blocksize, ULL Threadsize)
{
    LSH_Info* info = NULL;
    BYTE* test_pt = NULL;
    BYTE* sv_hashval = NULL;
    BYTE* us_cpu_pt = NULL;

    hipEvent_t start, stop;
    float elapsed_time_ms = 0.0f;

    info = (LSH_Info*)malloc(sizeof(LSH_Info));
    test_pt = (BYTE*)malloc(sizeof(BYTE) * TEST_PT_SIZE * Blocksize * Threadsize);
    sv_hashval = (BYTE*)malloc(sizeof(BYTE) * Blocksize * Threadsize * LSH_HASH_LEN);
    us_cpu_pt = (BYTE*)malloc(sizeof(BYTE) * TEST_PT_SIZE * Blocksize * Threadsize);

    int i, k = 0;

    for (i = 0; i < Blocksize * Threadsize; i++)
    {
        for (int j = 0; j < TEST_PT_SIZE; j++)
        {
            test_pt[TEST_PT_SIZE * i + j] = BYTE(j);
        }
    }

    BYTE* GPU_pt;
    BYTE* GPU_sv_hashval;
    LSH_Info* GPU_info;

    hipMalloc((void**)&GPU_pt, sizeof(BYTE) * TEST_PT_SIZE * Blocksize * Threadsize);
    hipMalloc((void**)&GPU_sv_hashval, sizeof(BYTE) * Blocksize * Threadsize * LSH_HASH_LEN);
    hipMalloc((void**)&GPU_info, sizeof(LSH_Info));

    for (i = 0; i < TEST_PT_SIZE; i++)
    {
        for (int j = 0; j < Blocksize * Threadsize; j++)
        {
            us_cpu_pt[k++] = test_pt[TEST_PT_SIZE * j + i];
        }
    }
    k = 0;

    printf("\n\nStart...\n");
    hipMemcpy(GPU_pt, us_cpu_pt, sizeof(BYTE) * TEST_PT_SIZE * Blocksize * Threadsize, hipMemcpyHostToDevice);
    hipMemcpy(GPU_info, info, sizeof(LSH_Info), hipMemcpyHostToDevice);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int x = 0; x < 1000; x++) {
        make_hash_val << <Blocksize, Threadsize >> > (GPU_info, GPU_pt, GPU_sv_hashval);
    }
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    hipMemcpy(sv_hashval, GPU_sv_hashval, sizeof(BYTE) * Blocksize * Threadsize * LSH_HASH_LEN, hipMemcpyDeviceToHost);
    elapsed_time_ms /= 1000;
    elapsed_time_ms = (Blocksize * Threadsize * LSH_BLOCK_LEN * sizeof(BYTE)) / elapsed_time_ms;
    elapsed_time_ms *= 1000;
    elapsed_time_ms /= (1024 * 1024 * 1024);
    printf("File size = %lld MB, Grid : %ld, Block : %ld, Performance : %4.2f GB/s\n", (Blocksize * Threadsize * LSH_BLOCK_LEN) / (1024 * 1024), Blocksize, Threadsize, elapsed_time_ms);
    getchar();
    getchar();

    hipGetLastError();
    hipDeviceSynchronize();


    printf("LSH_HAST_VAL : \n");
    for (i = 0; i < Blocksize * Threadsize * LSH_HASH_LEN; i++)
    {
        if (i % 32 == 0)
        {
            printf("\n%d��° hash��\n", k + 1);
        }
        printf(" %02X", sv_hashval[i]);
        if ((i + 1) % 8 == 0)
        {
            printf("\n");
        }
        if ((i + 1) % 32 == 0)
        {
            printf("\n");
            k++;
        }
    }

    return;
}

int main()
{
    ULL Blocksize = 1024, Threadsize = 128;

    test_LSH_GPU(Blocksize, Threadsize);

    return LSH_SUCCESS;
}